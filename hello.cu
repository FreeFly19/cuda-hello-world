
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1 << 29;

__global__ void vector_add(float *a, float *b, float *out, long n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	out[i] = a[i] + b[i];
}

int main(int argc, char **args) {
	float *a, *b, *out;
	float *d_a, *d_b, *d_out;

	a = (float*) malloc(sizeof(float) * N);
	b = (float*) malloc(sizeof(float) * N);
	out = (float*) malloc(sizeof(float) * N);
		
	hipMalloc((void**) &d_a, sizeof(float) * N);
	hipMalloc((void**) &d_b, sizeof(float) * N);
	hipMalloc((void**) &d_out, sizeof(float) * N);

	for(int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

	vector_add<<<N/1024, 1024>>>(d_a, d_b, d_out, N);
	
	hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

	//for(int i = 0; i < N; i++) {
	//	printf("%1.2f\n", out[i]);
	//}
	printf("%d", (int)out[N-1]/3);
}
